#include "hip/hip_runtime.h"
//
//  main.cpp
//  Parallel Degree of Separation
//
//  Created by Cary on 11/16/14.
//  Copyright (c) 2014 Cary. All rights reserved.
//

#include <iostream>
#include <fstream>
#include <cstdlib>
#include <map>
#include <vector>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>

#define limit 8000*8000
__global__ void relation(int *users, int *input_user1, int *input_user2,int *num_edge, int *num_node, int *base)
{
  int thid = threadIdx.x;
  int start_pos = blockIdx.x*blockDim.x;
  int user = users[thid+start_pos+*base];
  if(thid+start_pos+*base < *num_node)
  {
    for(int i = 0; i < *num_edge; i++)
    {
      if(input_user1[i] == user)
        input_user1[i] = thid+start_pos+*base;
      if(input_user2[i] == user)
        input_user2[i] = thid+start_pos+*base;   
     }
  }
}
__global__ void search_first_level(int *users, int *input_user1, int *input_user2, int *level_content, int *parent_content, int *friend_list, int *num_node, int *num_edge, int *num_friend, int *bound, int *offset, int *first_level, int *base)
{
  int thid = threadIdx.x;
  int start_pos = blockIdx.x*blockDim.x;
  int j = 0;
  if(thid+start_pos+*base < *num_node)
  {
    for(int i = 0; i < *num_edge; i++)
    {
      if(input_user1[i] == thid+start_pos+*base)
      {
//if(user == 5)
//{  printf("Find User 30!!!!!!! His Friend: %d\n ",input_user2[i]);
//}
        friend_list[(thid+start_pos)*(*num_node)+input_user2[i]] = input_user2[i];
        level_content[(thid+start_pos)*(*num_node)+j] = input_user2[i];
        first_level[offset[thid+start_pos+*base]+j] = input_user2[i];
        j++;
      }
      if(input_user2[i] == thid+start_pos+*base)
      {
//if(user == 5)
//{  printf("Find User 30!!!!!!! His Friend: %d\n ",input_user1[i]);
//}
 
        friend_list[(thid+start_pos)*(*num_node)+input_user1[i]] = input_user1[i];
        level_content[(thid+start_pos)*(*num_node)+j] = input_user1[i];
        first_level[offset[thid+start_pos+*base]+j] = input_user1[i];
        j++;
      }
    }
    num_friend[(thid+start_pos+*base)] = j;
    bound[(thid+start_pos+*base)] = j;
  }
}

__global__ void search_other_level(int *input_user1, int *input_user2, int *level_content, int *parent_content, int *friend_list, int *num_node, int *num_friend, int *in_bound_1, int *in_bound_2,int *offset, int *first_level, int *base, int *count)
{
  int thid = threadIdx.x;
  int start_pos = blockIdx.x*blockDim.x+*base;
  if(thid+start_pos-*base < *count)
  {
    int l_bound = in_bound_1[thid+start_pos];
    int u_bound = in_bound_2[thid+start_pos];
    in_bound_1[thid+start_pos] = u_bound;

    for(int k = l_bound; k < u_bound; k++)
    {
      int friend_t = level_content[(thid+start_pos-*base)*(*num_node)+k];  
      if(friend_t == -1)
      {
        break;
      }
      else
      {
        for(int n = 0; n < num_friend[friend_t]; n++)
        {
          int temp = first_level[offset[friend_t]+n];
          if((thid+start_pos-*base)*(*num_node)+temp > (*count)*(*num_node))
            printf("AAAAAAA %d %d %d %d %d\n",temp,(thid+start_pos-*base)*(*num_node)+temp,(*count)*(*num_node), thid+start_pos-*base,(*count) );
          if(friend_list[(thid+start_pos-*base)*(*num_node)+temp] == -1 && temp != thid+start_pos)
          {
            friend_list[(thid+start_pos-*base)*(*num_node)+temp] = temp;
            level_content[(thid+start_pos-*base)*(*num_node)+in_bound_2[thid+start_pos]] = temp;
            parent_content[(thid+start_pos-*base)*(*num_node)+in_bound_2[thid+start_pos]] = k;

            in_bound_2[thid+start_pos]++;
          }   
        } 
      }               
    }
  }
}

__global__ void find(int *user2,int *friend_list, int *level_content, int *parent_content, int *num_node, int *output, int *outsize, int *found, int *base)
{

  int thid = threadIdx.x;
  int start_pos = blockIdx.x*blockDim.x+*base;
  int parent_index = 0;
  if(thid+start_pos < *num_node)
  {

    if(level_content[(thid+start_pos)] == *user2  && friend_list[*user2] != -1 )
    {

      *found = 1;
      parent_index = parent_content[(thid+start_pos)];
      output[*outsize] = level_content[(thid+start_pos)];
      (*outsize)++;
      while(parent_index != -1)
      {
        output[*outsize] = level_content[parent_index];
        parent_index = parent_content[parent_index];
        (*outsize)++;
      }
    // printf("OUTSIZE %d\n", *outsize);
    }
  }
}

std::map<int, std::vector<int> > unique_user;
void addEdge(int user_id1, int user_id2) {
    std::map<int, std::vector<int> >::iterator it;    
    // Insert edge into user_id1's firend list
    it = unique_user.find(user_id1);
    if (it == unique_user.end()) {
        std::vector<int> friend_list(1, user_id2);
        unique_user.insert(std::pair<int, std::vector<int> >(user_id1, friend_list));
    } else {
        it->second.push_back(user_id2);
    }

    // Insert edge into id2's firend list
    it = unique_user.find(user_id2);
    if (it == unique_user.end()) {
        std::vector<int> friend_list(1, user_id1);
        unique_user.insert(std::pair<int, std::vector<int> >(user_id2, friend_list));
    } else {
        it->second.push_back(user_id1);
    }    
}

int main(int argc, const char * argv[]) {
    if (argc < 2) {
        std::cout<<"Please enter the path of input data file\n";
        return 0;
    }

    struct timeval starttime,endtime;

    thrust::device_vector<int> user1;
    thrust::device_vector<int> user2;


    gettimeofday(&starttime,NULL);
    
    //reading file
    std::ifstream data_file;
    data_file.open(argv[1]);
    if (data_file.is_open()) {
        //cout<<"File opened"<<endl;
      int id1, id2;
      while (data_file.eof() == false) {
        data_file>>id1>>id2;
        user1.push_back(id1);
        user2.push_back(id2);
        addEdge(id1,id2);
      }
    } else {
        std::cout<<"File did not open\n";
        exit(0);
    }
    data_file.close();
    
    int num_node = unique_user.size();
    int num_edge = user1.size();

    std::cout<<"#Node := "<<num_node<<"\n";
    std::cout<<"#Edge := "<<num_edge<<"\n";

    int *offset = new int [num_node];
    int *users = new int[num_node];

    std::map<int, std::vector<int> >::iterator it;    
    int ttt = 0;
    for(it = unique_user.begin(); it != unique_user.end(); ++it)
    {
      users[ttt] = it->first;
      offset[ttt] = it->second.size();
      ttt++;
    }
    if(ttt != num_node)
      std::cout<<"ERROR!!!!! \n";
    int add = offset[num_node-1];
    thrust::exclusive_scan(offset, offset + num_node , offset); // in-place scan

    gettimeofday(&endtime,NULL);
    long long time = ((endtime.tv_sec * 1000000 + endtime.tv_usec) - (starttime.tv_sec * 1000000 + starttime.tv_usec));
    printf(">>>Initializaton takes := %lld microseconds \n\n", time);

    int search_depth;
    std::cout<<"#Level you want to search: ";
    std::cin>>search_depth;

    int num_threads = limit/num_node;
    int block_size = num_threads > 512 ? 512 : num_threads;
    int num_blocks = ceil(num_threads/block_size);

    std::cout<<"#threads: "<<num_threads<<" #Blocks: "<<num_blocks<<"\n";

    dim3 dimGrid(num_blocks);
    dim3 dimBlock(block_size);

    float para_time;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 
    hipEventRecord(start);
    long long num_num = (long long)num_node*num_node;
    int *host_level_content = new int [num_num];
    int *host_parent_content = new int [num_num];
    int *host_friend_list = new int [num_num];


    int *device_user1 = thrust::raw_pointer_cast(&user1[0]);
    int *device_user2 = thrust::raw_pointer_cast(&user2[0]);
    int *device_offset;
    int *device_users;
    int *device_level_content;
    int *device_parent_content;
    int *device_num_friend;
    int *friend_list; 
    int *device_num_node;
    int *device_num_edge;
    int *inbound_1;
    int *inbound_2;
    int *device_first_level;
    int *device_base;
    int *device_count;

    hipError_t error; 

    hipMalloc((int**)&device_num_friend,num_node*sizeof(int));
    hipMalloc((int**)&device_offset,num_node*sizeof(int));
    hipMalloc((int**)&device_users,num_node*sizeof(int));
    hipMalloc((int**)&device_first_level,(offset[num_node-1]+add)*sizeof(int));
    hipMalloc((int**)&device_num_node,sizeof(int));
    hipMalloc((int**)&device_num_edge,sizeof(int));
    hipMalloc((int**)&inbound_1,num_node*sizeof(int));
    hipMalloc((int**)&inbound_2,num_node*sizeof(int));
    hipMalloc((int**)&device_base,sizeof(int));
    hipMalloc((int**)&device_count,sizeof(int));

    hipMemset((int**)inbound_1,0,num_node*4);
    hipMemcpy(device_num_node, &num_node, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_num_edge, &num_edge, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_offset, offset, num_node*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_users, users, num_node*sizeof(int), hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
      printf("CUDA first error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
    long long count = 0;
    long long base = 0;    
    count = limit/num_node; 
    if(count > num_node)
      count = num_node;
    do
    {    
      hipMemcpy(device_base, &base, sizeof(int), hipMemcpyHostToDevice);
      relation<<<dimGrid,dimBlock>>>(device_users, device_user1, device_user2, device_num_edge, device_num_node, device_base);
      base = base + count;
      if(base+count > num_node)
        count = num_node - base;
    }while(base < num_node);


    base = 0;   
    count = limit/num_node; 
    if(count > num_node)
      count = num_node;
    do{
      hipMalloc((int**)&device_level_content,count*num_node*sizeof(int));
      hipMalloc((int**)&device_parent_content,count*num_node*sizeof(int));
      hipMalloc((int**)&friend_list,count*num_node*sizeof(int));

      error = hipGetLastError();    
      if(error != hipSuccess)
      {
        printf("CUDA second error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      hipMemset((int**)device_level_content,-1,count*num_node*4);
      hipMemset((int**)device_parent_content,-1,count*num_node*4);
      hipMemset((int**)friend_list,-1,count*num_node*4);
      hipMemcpy(device_base, &base, sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(device_count, &count, sizeof(int), hipMemcpyHostToDevice);
      error = hipGetLastError();    
      if(error != hipSuccess)
      {
        printf("CUDA third error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      search_first_level<<<dimGrid,dimBlock>>>(device_users,device_user1, device_user2, device_level_content, device_parent_content, friend_list, device_num_node, device_num_edge, device_num_friend,inbound_2, device_offset, device_first_level,device_base);

      for(int i = 1; i < search_depth; i++)
      {      
        search_other_level<<<dimGrid,dimBlock>>>(device_user1, device_user2, device_level_content, device_parent_content, friend_list, device_num_node, device_num_friend, inbound_1, inbound_2,device_offset, device_first_level,device_base,device_count);
      } 
      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA search_level error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      hipMemcpy((host_level_content+base*num_node), device_level_content, count*num_node*4, hipMemcpyDeviceToHost);
      hipMemcpy((host_parent_content+base*num_node), device_parent_content, count*num_node*4, hipMemcpyDeviceToHost);
    //  printf("BASE: %d Count %d Node %d\n",base, count, num_node);
    //  if(base*num_node + count*num_node > num_node*num_node)
    //    printf("OUT OF BOUND!!!!!!!  %d %d %d %d\n",base*num_node,count*num_node,base*num_node + count*num_node,num_node*num_node);
      hipMemcpy((host_friend_list+base*num_node), friend_list, count*num_node*4, hipMemcpyDeviceToHost);
      hipFree(device_level_content);
      hipFree(device_parent_content);
      hipFree(friend_list);
      base = base + count;
      if(base+count > num_node)
        count = num_node - base;

    }while(base < num_node);

    hipFree(device_num_edge);
    hipFree(device_num_friend);
    hipFree(inbound_1);
    hipFree(inbound_2);
    hipFree(device_base);
    hipFree(device_count);
    hipFree(device_offset);
    hipFree(device_users);

    error = hipGetLastError();
    if(error != hipSuccess)
    {
      printf("CUDA first free error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&para_time,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("\nParallel Search Time:%f microseconds",para_time);


    bool should_continue = true;
    while (should_continue) {
      int user_id1, user_id2;
      std::map<int, std::vector<int> >::iterator it;
      do
      {
        std::cout<<"\n";
        std::cout<<"Enter first user's ID: ";
        std::cin >> user_id1;
        it = unique_user.find(user_id1);
        if(it == unique_user.end())
          std::cout << "User Not Exist\n";
      }
      while(it == unique_user.end());
     
      do
      {
        std::cout<<"\n";
        std::cout<<"Enter second user's ID: ";
        std::cin >> user_id2;
        it = unique_user.find(user_id2);
        if(it == unique_user.end())
          std::cout << "User Not Exist\n";
      }
      while(it == unique_user.end());
    
      int base_base;
      int base_base_1;
      int base_found = 0;
      int base_found_1 = 0;
      for(int i = 0; i < num_node; i++)
      {
        if(users[i] == user_id1)
        {
          base_base = i;
          base_found = 1;
        }
        if(users[i] == user_id2)
        {
          base_base_1 = i;
          base_found_1 = 1;
        }
        if(base_found == 1 && base_found_1 == 1)
          break;
      }
     // printf("BASEBASE %d %d\n", base_base, users[base_base]);
      int* device_input2;
      int* device_output;
      int* device_size;
      int* found;
      int *found_host = new int;
      int *size = new int;
      int *result = new int[*size];

      hipMalloc((int**)&device_level_content,num_node*sizeof(int));
      hipMalloc((int**)&device_parent_content,num_node*sizeof(int));
      hipMalloc((int**)&friend_list,num_node*sizeof(int));
      hipMalloc((int**)&device_input2,sizeof(int));
      hipMalloc((int**)&device_size,sizeof(int));
      hipMalloc((int**)&device_output,search_depth*sizeof(int));
      hipMalloc((int**)&found,sizeof(int));
      hipMalloc((int**)&device_base,sizeof(int));
      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA second malloc error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      hipMemset((int**)device_output,0,search_depth*4);
      hipMemset((int**)device_size,0,4);
      hipMemset((int**)found,0,4);
      hipMemcpy(device_input2, &base_base_1, sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(device_level_content, host_level_content+base_base*num_node, num_node*sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(device_parent_content, host_parent_content+base_base*num_node, num_node*sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(friend_list, host_friend_list+base_base*num_node, num_node*sizeof(int), hipMemcpyHostToDevice);
      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA second memset memcpy error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      base = 0;
      count = limit/num_node; 
      if(count > num_node)
        count = num_node;
      do{
        
        hipMemcpy(device_base, &base, sizeof(int), hipMemcpyHostToDevice);
        find<<<dimGrid, dimBlock>>>(device_input2, friend_list,device_level_content, device_parent_content, device_num_node, device_output, device_size,found,device_base);
        hipMemcpy(found_host, found, sizeof(int), hipMemcpyDeviceToHost);

        base = base + count;
        if(base+count > num_node)
          count = num_node - base;
      }while(base < num_node && !(*found_host));
      
      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA find error: %s\n", hipGetErrorString(error));
        exit(-1);
      }


      hipMemcpy(size, device_size, sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(result, device_output, (*size)*sizeof(int), hipMemcpyDeviceToHost);

      //printf("SIZE %d\n",*size);

      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA find error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      hipFree(device_input2);
      hipFree(device_size);
      hipFree(device_output);
      hipFree(found);
      hipFree(device_level_content);
      hipFree(device_parent_content);
      hipFree(device_base);

      if(*found_host == 0)
        std::cout<<"\nUser "<<user_id1<<" and User "<<user_id2<<" are not connected within search level := "<<search_depth<<"\n\n";
      else
      {
        std::cout<<"\nDegree of Separation between User "<<user_id1<<" and User "<<user_id2<<" is "<<(*size)<<"\n\n";
        for (int i = 0; i < (*size); i++) {
          std::cout<<users[result[i]]<<"<-";
        }      
        std::cout<<user_id1<<"\n";
      }

      std::cout<<"\nDo you want to continue? 1 for yes and 0 for no: ";
      std::cin>>should_continue;
      delete[] result;
      delete size;
      delete found_host;
    }

   delete[] offset;
   delete[] users;
   delete[] host_level_content;
   delete[] host_parent_content;
   delete[] host_friend_list;


    return 0;
}
