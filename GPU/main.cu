#include "hip/hip_runtime.h"
//
//  main.cpp
//  Parallel Degree of Separation
//
//  Created by Cary on 11/16/14.
//  Copyright (c) 2014 Cary. All rights reserved.
//

#include <iostream>
#include <fstream>
#include <cstdlib>
#include <set>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__global__ void search_first_level(int *input_user1, int *input_user2, int *level_content, int *parent_content, int *friend_list, int *num_node, int *num_edge, int *num_friend, int *bound)
{
  int thid = threadIdx.x;
  int start_pos = blockIdx.x*blockDim.x;
  int j = 0;
  if(thid+start_pos < *num_node)
  {
    
    for(int i = 0; i < *num_edge; i++)
    {
      if(input_user1[i] == (thid+start_pos))
      {
        friend_list[(thid+start_pos)*(*num_node)+input_user2[i]] = input_user2[i];
        level_content[(thid+start_pos)*(*num_node)+j] = input_user2[i];
        j++;
      }
      if(input_user2[i] == (thid+start_pos))
      {
        friend_list[(thid+start_pos)*(*num_node)+input_user1[i]] = input_user1[i];
        level_content[(thid+start_pos)*(*num_node)+j] = input_user1[i];
        j++;
      }
    }
    num_friend[(thid+start_pos)] = j;
    bound[(thid+start_pos)] = j;
  }
}

__global__ void search_other_level(int *input_user1, int *input_user2, int *level_content, int *parent_content, int *friend_list, int *num_node, int *num_friend, int *in_bound_1, int *in_bound_2)
{
  int thid = threadIdx.x;
  int start_pos = blockIdx.x*blockDim.x;
  if(thid+start_pos < *num_node)
  {
    int l_bound = in_bound_1[thid+start_pos];
    int u_bound = in_bound_2[thid+start_pos];
    in_bound_1[thid+start_pos] = u_bound;
    for(int k = l_bound; k < u_bound; k++)
    {
      int friend_t = level_content[(thid+start_pos)*(*num_node)+k];  
      if(friend_t == -1)
      {
        break;
      }
      else
      {
        for(int n = 0; n < num_friend[friend_t]; n++)
        {
          int temp = level_content[friend_t*(*num_node)+n];
          if((thid+start_pos)*(*num_node)+temp > (*num_node)*(*num_node))
            printf("friend_t: %d num_friend: %d\n", friend_t,num_friend[friend_t]);
          if(friend_list[(thid+start_pos)*(*num_node)+temp] == -1)
          {
            friend_list[(thid+start_pos)*(*num_node)+temp] = temp;
            level_content[(thid+start_pos)*(*num_node)+in_bound_2[thid+start_pos]] = temp;
            parent_content[(thid+start_pos)*(*num_node)+in_bound_2[thid+start_pos]] =(thid+start_pos)*(*num_node)+k;
            in_bound_2[thid+start_pos]++;
          }   
        } 
      }               
    }
  }
}

__global__ void find(int *user1, int *user2, int *friend_list, int *level_content, int *parent_content, int *num_node, int *output, int *outsize, int *found)
{

  int thid = threadIdx.x;
  int start_pos = blockIdx.x*blockDim.x;
  int parent_index = 0;
  *outsize = 0;
  if(thid == 0 && start_pos == 0)
  {
    *found = 0;
  }
  if(thid+start_pos < *num_node)
  {
    if(level_content[(*user1)*(*num_node)+(thid+start_pos)] == *user2)
    {
      *found = 1;
      parent_index = parent_content[(*user1)*(*num_node)+(thid+start_pos)];
      output[*outsize] = level_content[(*user1)*(*num_node)+(thid+start_pos)];
      (*outsize)++;
      while(parent_index != -1)
      {
        output[*outsize] = level_content[parent_index];
        parent_index = parent_content[parent_index];
        (*outsize)++;
      }
      // printf("Parent index: %d\n, Parent: %d thread: %d\n",parent_index, level_content[parent_index], thid+start_pos);
    }
  }
}

int main(int argc, const char * argv[]) {
    if (argc < 2) {
        std::cout<<"Please enter the path of input data file\n";
        return 0;
    }

    struct timeval starttime,endtime;

    thrust::device_vector<int> user1;
    thrust::device_vector<int> user2;
    std::set<int> unique_user;

    gettimeofday(&starttime,NULL);
    
    //reading file
    std::ifstream data_file;
    data_file.open(argv[1]);
    if (data_file.is_open()) {
        //cout<<"File opened"<<endl;
        int id1, id2;
        while (data_file.eof() == false) {
            data_file>>id1>>id2;
	    user1.push_back(id1);
            user2.push_back(id2);
            unique_user.insert(id1);
            unique_user.insert(id2);
        }
    } else {
        std::cout<<"File did not open\n";
        exit(0);
    }
    data_file.close();
    
    int num_node = unique_user.size();
    int num_edge = user1.size();
    std::cout<<"#Node := "<<num_node<<"\n";
    std::cout<<"#Edge := "<<num_edge<<"\n";
 
    gettimeofday(&endtime,NULL);
    long long time = ((endtime.tv_sec * 1000000 + endtime.tv_usec) - (starttime.tv_sec * 1000000 + starttime.tv_usec));
    printf(">>>Initializaton takes := %lld microseconds \n\n", time);

    int search_depth;
    std::cout<<"#Level you want to search: ";
    std::cin>>search_depth;

    int num_threads = num_node;
    int num_blocks = ceil(num_threads/512.0);

    std::cout<<"#threads: "<<num_threads<<" #Blocks: "<<num_blocks<<"\n";

    dim3 dimGrid(num_blocks);
    dim3 dimBlock(512);

    float para_time;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 
    hipEventRecord(start);

    int *device_user1 = thrust::raw_pointer_cast(&user1[0]);
    int *device_user2 = thrust::raw_pointer_cast(&user2[0]);
    int *device_level_content;
    int *device_parent_content;
    int *device_num_friend;
    int *friend_list; 
    int *device_num_node;
    int *device_num_edge;
    int *inbound_1;
    int *inbound_2;

    hipError_t error; 

    hipMalloc((int**)&device_level_content,num_node*num_node*sizeof(int));
    hipMalloc((int**)&device_parent_content,num_node*num_node*sizeof(int));
    hipMalloc((int**)&friend_list,num_node*num_node*sizeof(int));
    hipMalloc((int**)&device_num_friend,num_node*sizeof(int));
    hipMalloc((int**)&device_num_node,sizeof(int));
    hipMalloc((int**)&device_num_edge,sizeof(int));
    hipMalloc((int**)&inbound_1,num_node*sizeof(int));
    hipMalloc((int**)&inbound_2,num_node*sizeof(int));

    error = hipGetLastError();
    if(error != hipSuccess)
    {
      printf("CUDA first malloc error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    hipMemset((int**)device_level_content,-1,num_node*num_node);
    hipMemset((int**)device_parent_content,-1,num_node*num_node);
    hipMemset((int**)friend_list,-1,num_node*num_node);
    hipMemset((int**)inbound_1,0,num_node);
    hipMemcpy(device_num_node, &num_node, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_num_edge, &num_edge, sizeof(int), hipMemcpyHostToDevice);

    error = hipGetLastError();    
    if(error != hipSuccess)
    {
      printf("CUDA first memset memcpy error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    search_first_level<<<dimGrid,dimBlock>>>(device_user1, device_user2, device_level_content, device_parent_content, friend_list, device_num_node, device_num_edge, device_num_friend,inbound_2);

    for(int i = 1; i < search_depth; i++)
    {      
      search_other_level<<<dimGrid,dimBlock>>>(device_user1, device_user2, device_level_content, device_parent_content, friend_list, device_num_node, device_num_friend, inbound_1, inbound_2);
    } 
    error = hipGetLastError();
    if(error != hipSuccess)
    {
      printf("CUDA search_level error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    hipFree(device_num_edge);
    hipFree(device_num_friend);
    hipFree(inbound_1);
    hipFree(inbound_2);

    error = hipGetLastError();
    if(error != hipSuccess)
    {
      printf("CUDA first free error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&para_time,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("\nParallel Search Time:%f microseconds",para_time);


    bool should_continue = true;
    while (should_continue) {
      int user_id1, user_id2;
      std::set<int>::iterator it;
      do
      {
        std::cout<<"\n";
        std::cout<<"Enter first user's ID: ";
        std::cin >> user_id1;
        it = unique_user.find(user_id1);
        if(it == unique_user.end())
          std::cout << "User Not Exist\n";
      }
      while(it == unique_user.end());
     
      do
      {
        std::cout<<"\n";
        std::cout<<"Enter second user's ID: ";
        std::cin >> user_id2;
        it = unique_user.find(user_id2);
        if(it == unique_user.end())
          std::cout << "User Not Exist\n";
      }
      while(it == unique_user.end());

      int* device_input1;
      int* device_input2;
      int* device_output;
      int* device_size;
      int* found;

     
      hipMalloc((int**)&device_input1,sizeof(int));
      hipMalloc((int**)&device_input2,sizeof(int));
      hipMalloc((int**)&device_size,sizeof(int));
      hipMalloc((int**)&device_output,search_depth*sizeof(int));
      hipMalloc((int**)&found,sizeof(int));
      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA second malloc error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      hipMemset((int**)device_output,0,search_depth);
      hipMemset((int**)device_size,0,1);
      hipMemcpy(device_input1, &user_id1, sizeof(int), hipMemcpyHostToDevice);
      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA second memset memcpy error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      hipMemcpy(device_input2, &user_id2, sizeof(int), hipMemcpyHostToDevice);



      find<<<dimGrid, dimBlock>>>(device_input1, device_input2, friend_list,device_level_content, device_parent_content, device_num_node, device_output, device_size,found);
      error = hipGetLastError();
      if(error != hipSuccess)
      {
        printf("CUDA find error: %s\n", hipGetErrorString(error));
        exit(-1);
      }

      int* found_host = new int;
      int* size = new int;
      hipMemcpy(found_host, found, sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(size, device_size, sizeof(int), hipMemcpyDeviceToHost);

      int* result = new int[*size];
      hipMemcpy(result, device_output, (*size)*sizeof(int), hipMemcpyDeviceToHost);

      hipFree(device_input1);
      hipFree(device_input2);
      hipFree(device_size);
      hipFree(device_output);
      hipFree(found);

      if(*found_host == 0)
        std::cout<<"\nUser "<<user_id1<<" and User "<<user_id2<<" are not connected within search level := "<<search_depth<<"\n\n";
      else
      {
        std::cout<<"\nDegree of Separation between User "<<user_id1<<" and User "<<user_id2<<" is "<<(*size)<<"\n\n";
        for (int i = 0; i < (*size); i++) {
          std::cout<<result[i]<<"<-";
        }      
        std::cout<<user_id1<<"\n";
      }

      std::cout<<"\nDo you want to continue? 1 for yes and 0 for no: ";
      std::cin>>should_continue;
    }

   
    hipFree(device_level_content);
    hipFree(device_parent_content);
    hipFree(friend_list);
    hipFree(device_num_node);


    return 0;
}
